#include "HashTableDevice.h"
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"
#include <hip/hip_runtime.h>

HashTableDevice::HashTableDevice(void)
{
}

HashTableDevice::~HashTableDevice(void)
{
}

void HashTableDevice::init(int number_particles, int number_cells)
{
	hipMalloc((void **)&_gridHash, number_particles * sizeof(unsigned int));
	hipMalloc((void **)&_gridIndex, number_particles * sizeof(unsigned int));
	hipMalloc((void **)&_cellStart, number_cells * sizeof(unsigned int));
	hipMalloc((void **)&_cellEnd, number_cells * sizeof(unsigned int));

	// radix sort
	//hipMalloc((void **)&_radixHash, number_particles*sizeof(unsigned int));
	//hipMalloc((void **)&_radixIndex, number_particles*sizeof(unsigned int));
}

void HashTableDevice::free(void)
{
	hipFree(_gridHash);
	hipFree(_gridIndex);
	hipFree(_cellStart);
	hipFree(_cellEnd);
	//hipFree(_radixHash);
	//hipFree(_radixIndex);
}


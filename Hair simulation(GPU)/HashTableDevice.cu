#include "HashTableDevice.h"
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"
#include <hip/hip_runtime.h>

HashTableDevice::HashTableDevice(void)
{
}

HashTableDevice::~HashTableDevice(void)
{
}

void HashTableDevice::init(int num_particles, int num_cells)
{
	hipMalloc((void **)&_gridParticleHash, num_particles * sizeof(uint));
	hipMalloc((void **)&_gridParticleIndex, num_particles * sizeof(uint));
	hipMalloc((void**)&_cellStart, num_cells * sizeof(uint));
	hipMalloc((void**)&_cellEnd, num_cells * sizeof(uint));
	printf("init hashing\n");
}

void HashTableDevice::free(void)
{
	hipFree(_gridParticleHash);
	hipFree(_gridParticleIndex);
	hipFree(_cellStart);
	hipFree(_cellEnd);
}
#include "hip/hip_runtime.h"
﻿#pragma once
#include <stdio.h>
#include "HairModel.h"
#include <hip/hip_runtime.h>
#include "" 
#include "vector_calc.cuh"


void HairModel::device_init() {
	hipMalloc((void**)&p_i_d,sizeof(int) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&p_p_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&p_v_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&p_f_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&r_p_p_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&s_p_p_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	
	//hipMalloc((void**)&r_s_p_p,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&s_f_d,sizeof(Frame) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&t_d,sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE);
	hipMalloc((void**)&r_p_l_d,sizeof(double) * STRAND_SIZE * PARTICLE_SIZE);
	
	hipMemcpy(p_i_d, p_i, sizeof(int) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(p_p_d, p_p, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(r_p_p_d, r_p_p, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	//hipMemcpy(r_s_f, h.r_s_f, sizeof(Frame) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	//hipMemcpy(r_s_p_p, h.r_s_p_p, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(t_d, t, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(r_p_l_d, r_p_l, sizeof(double) * STRAND_SIZE, hipMemcpyHostToDevice);


	array_init << <STRAND_SIZE, PARTICLE_SIZE >> > (p_f_d);
	array_init << <STRAND_SIZE, PARTICLE_SIZE >> > (p_v_d);
}

//__global__ void integrate_internal_hair_force(int *p_i,double3 *p_p, double3 *r_p_p, Frame *s_f, double3* _t,double3 *p_f, double3 *p_v) {
//	double dt = 0.0009;
//	
//	for (int i = 0; i < PARTICLE_SIZE; i++) {
//		int tid = p_i[threadIdx.x * PARTICLE_SIZE + i];
//		if (i < PARTICLE_SIZE - 1) {
//			//Stretch spring
//			double3 e = vector_sub_k(p_p[tid + 1], p_p[tid]);
//			double3 rest_e = vector_sub_k(r_p_p[tid + 1], r_p_p[tid]);
//			double3 e_hat = vector_normalized_k(e);
//
//			double3 force = vector_multiply_k(e_hat, (vector_length_k(e) - vector_length_k(rest_e)) * K_S);
//
//			p_f[tid] = vector_add_k(p_f[tid], force);
//			p_f[tid + 1] = vector_sub_k(p_f[tid + 1], force);
//
//			//Bending spring
//			double3 t = multiply_frame_k(s_f[tid - 1], _t[tid]);
//			force = vector_multiply_k(vector_sub_k(e, t), K_B);
//			
//			p_f[tid] = vector_add_k(p_f[tid], force);
//			p_f[tid + 1] = vector_sub_k(p_f[tid + 1], force);
//			
//
//		}
//		if (i > 0) {
//			double3 ac = p_f[tid];
//			p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
//			p_f[tid] = make_double3(0.0, 0.0, 0.0);
//		}
//	}
//}

__global__ void integrate(double3 *p_p, double3 *p_f, double3 *p_v, double dt) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x > 0) {
		double3 ac = p_f[tid];
		p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
		p_f[tid] = make_double3(0.0, 0.0, 0.0);
	}
}

__global__ void integrate_internal_hair_force(double3 *p_p, double3 *r_p_p, Frame *s_f, double3* _t, double3 *p_f, double3 *p_v) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	double dt = 0.0009;

	//if(blockIdx.x == 0)printf("thread: %d\n", threadIdx.x);
	double3 e = vector_sub_k(p_p[tid + 1], p_p[tid]);
	double3 rest_e = vector_sub_k(r_p_p[tid + 1], r_p_p[tid]);
	double3 e_hat = vector_normalized_k(e);

	double3 force1 = vector_multiply_k(e_hat,(vector_length_k(e)-vector_length_k(rest_e)) * K_S);

	double3 t = multiply_frame_k(s_f[tid - 1], _t[tid]);
	double3 force2 = vector_multiply_k(vector_sub_k(e, t), K_B);
		
	double3 result = vector_add_k(force1, force2);
	p_f[tid] = vector_add_k(p_f[tid], result);
	__syncthreads();
	p_f[tid + 1] = vector_sub_k(p_f[tid + 1], result);
	
}



//__global__ void integrate_external_hair_force(int *p_i, double3 *p_p, double3 *p_f, double3 *p_v) {
//	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	double dt = 0.0009;
//	double3 gravity = make_double3(0.0, -10, 0.0);
//	for (int i = 0; i < PARTICLE_SIZE; i++) {
//		int tid = p_i[threadIdx.x * PARTICLE_SIZE + i];
//		if (i < PARTICLE_SIZE - 1) {
//			p_f[tid] = vector_add_k(p_f[tid], gravity);
//		}
//		if (i > 0) {
//			double3 ac = p_f[tid];
//			p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
//			p_f[tid] = make_double3(0.0, 0.0, 0.0);
//		}
//	}
//}

__global__ void integrate_external_hair_force(double3 *p_p, double3 *p_f, double3 *p_v) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	double dt = 0.0009;
	double3 gravity = make_double3(0.0, -10, 0.0);
	if (threadIdx.x < PARTICLE_SIZE - 1) {
		p_f[tid] = vector_add_k(p_f[tid], gravity);
	}

	//if (threadIdx.x > 0) {
	//	double3 ac = p_f[tid];
	//	p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
	//	p_f[tid] = make_double3(0.0, 0.0, 0.0);
	//}
}

//__global__ void integrate_damping_force(int *p_i, double3 *p_p, double3 *p_f, double3 *p_v) {
//	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	double dt = 0.00009;
//
//	for (int i = 0; i < PARTICLE_SIZE; i++) {
//		int tid = p_i[threadIdx.x * PARTICLE_SIZE + i];
//		if (i < PARTICLE_SIZE - 1) {
//			//Stretch damping
//			double3 d_v = vector_sub_k(p_v[tid + 1], p_v[tid]);
//			double3 e = vector_sub_k(p_p[tid + 1], p_p[tid]);
//			double3 e_hat = vector_normalized_k(e);
//
//			double3 force = vector_multiply_k(vector_multiply_k(e_hat, vector_dot_k(d_v, e_hat)), C_S);
//			p_f[tid] = vector_add_k(p_f[tid], force);
//			p_f[tid + 1] = vector_sub_k(p_f[tid + 1], force);
//
//			//Bending damping
//			force = vector_multiply_k(vector_sub_k(d_v, vector_multiply_k(e_hat, vector_dot_k(d_v, e_hat))), C_B);
//			p_f[tid] = vector_add_k(p_f[tid], force);
//			p_f[tid + 1] = vector_sub_k(p_f[tid + 1], force);
//		}
//
//		if (i > 0) {
//			double3 ac = p_f[tid];
//			p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
//			p_f[tid] = make_double3(0.0, 0.0, 0.0);
//		}
//	}
//}

__global__ void integrate_damping_force(double3 *p_p, double3 *p_f, double3 *p_v) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	double dt = 0.00009;

	if (threadIdx.x < PARTICLE_SIZE - 1) {
		
		double3 d_v = vector_sub_k(p_v[tid + 1], p_v[tid]);
		double3 e = vector_sub_k(p_p[tid + 1], p_p[tid]);
		double3 e_hat = vector_normalized_k(e);

		double3 force1 = vector_multiply_k(vector_multiply_k(e_hat, vector_dot_k(d_v, e_hat)), C_S);
		double3 force2 = vector_multiply_k(vector_sub_k(d_v, vector_multiply_k(e_hat, vector_dot_k(d_v, e_hat))), C_B);

		double3 result = vector_add_k(force1, force2);
		p_f[tid] = vector_add_k(p_f[tid], result);
		__syncthreads();
		p_f[tid + 1] = vector_sub_k(p_f[tid + 1], result);

	}

	//if (threadIdx.x > 0) {
	//	double3 ac = p_f[tid];
	//	p_v[tid] = vector_add_k(p_v[tid], vector_multiply_k(ac, dt));
	//	p_f[tid] = make_double3(0.0, 0.0, 0.0);
	//}
}

//__global__ void update_position(int *p_i, double3 *p_p, double3 *p_v) {
//	double dt = 0.01;
//	for (int i = 1; i < PARTICLE_SIZE; i++) {
//		int tid = p_i[threadIdx.x * PARTICLE_SIZE + i];
//		p_p[tid] = vector_add_k(p_p[tid], vector_multiply_k(p_v[tid], dt));
//	}
//}


__global__ void update_position(double3 *p_p, double3 *p_v) {
	double dt = 0.01;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x > 0) {
		p_p[tid] = vector_add_k(p_p[tid], vector_multiply_k(p_v[tid], dt));
	}
}

void HairModel:: simulation() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int iter1 = 0; iter1 < 2; iter1++) {
		for (int iter2 = 0; iter2 < 15; iter2++) {
			//hipMemcpy(p_p_d, s_p_p_d, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyDeviceToDevice);
			//smoothing_function_k<< <1, STRAND_SIZE >> > (p_p_d, s_p_p_d, r_p_l_d, 0.23, true);
			//compute_frame_k << <1, STRAND_SIZE >> > (s_f_d, s_p_p_d);

			integrate_internal_hair_force <<<STRAND_SIZE, PARTICLE_SIZE>> > (p_p_d, r_p_p_d, s_f_d, t_d , p_f_d, p_v_d);
			integrate << <STRAND_SIZE, PARTICLE_SIZE >> > (p_p_d, p_f_d, p_v_d, 0.0009);

			integrate_external_hair_force <<<STRAND_SIZE, PARTICLE_SIZE >> > (p_p_d, p_f_d, p_v_d);
			integrate << <STRAND_SIZE, PARTICLE_SIZE >> > (p_p_d, p_f_d, p_v_d, 0.0009);

			for (int iter3 = 0; iter3 < 10 * iter2; iter3++) {
				integrate_damping_force <<<STRAND_SIZE, PARTICLE_SIZE>> > (p_p_d, p_f_d, p_v_d);
				integrate << <STRAND_SIZE, PARTICLE_SIZE >> > (p_p_d, p_f_d, p_v_d, 0.00009);
			}
		}
		update_position <<<STRAND_SIZE, PARTICLE_SIZE >> > (p_p_d, p_v_d);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << " SAXPY execution time : " << milliseconds << " ms " << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(p_p, p_p_d, sizeof(double3) * STRAND_SIZE * PARTICLE_SIZE, hipMemcpyDeviceToHost);
}

